
#include <hip/hip_runtime.h>
// REQUIRES: clang-driver
// REQUIRES: x86-registered-target
// REQUIRES: nvptx-registered-target

// -flto=thin causes a switch to llvm-bc object files.
// RUN: %clangxx -ccc-print-phases -nocudainc -nocudalib -c %s -flto=thin 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILE-ACTIONS < %t %s
//
// CHECK-COMPILE-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILE-ACTIONS-NOT: lto-bc
// CHECK-COMPILE-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)

// RUN: %clangxx -ccc-print-phases -nocudainc -nocudalib %s -flto=thin 2> %t
// RUN: FileCheck -check-prefix=CHECK-COMPILELINK-ACTIONS < %t %s
//
// CHECK-COMPILELINK-ACTIONS: 0: input, "{{.*}}thinlto.cu", cuda, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 1: preprocessor, {0}, cuda-cpp-output
// CHECK-COMPILELINK-ACTIONS: 2: compiler, {1}, ir, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 3: input, "{{.*}}thinlto.cu", cuda, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 4: preprocessor, {3}, cuda-cpp-output, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 5: compiler, {4}, ir, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 6: backend, {5}, assembler, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 7: assembler, {6}, object, (device-cuda, sm_20)
// CHECK-COMPILELINK-ACTIONS: 8: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_20)" {7}, object
// CHECK-COMPILELINK-ACTIONS: 9: offload, "device-cuda (nvptx{{.*}}-nvidia-cuda:sm_20)" {6}, assembler
// CHECK-COMPILELINK-ACTIONS: 10: linker, {8, 9}, cuda-fatbin, (device-cuda)
// CHECK-COMPILELINK-ACTIONS: 11: offload, "host-cuda {{.*}}" {2}, "device-cuda{{.*}}" {10}, ir
// CHECK-COMPILELINK-ACTIONS: 12: backend, {11}, lto-bc, (host-cuda)
// CHECK-COMPILELINK-ACTIONS: 13: linker, {12}, image, (host-cuda)

// -flto=thin should cause link using gold plugin with thinlto option,
// also confirm that it takes precedence over earlier -fno-lto and -flto=full.
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:        -target x86_64-unknown-linux -### %s -flto=full -fno-lto -flto=thin 2> %t
// RUN: FileCheck -check-prefix=CHECK-LINK-THIN-ACTION < %t %s
//
// CHECK-LINK-THIN-ACTION: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"
// CHECK-LINK-THIN-ACTION: "-plugin-opt=thinlto"

// Check that subsequent -flto=full takes precedence
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:        -target x86_64-unknown-linux -### %s -flto=thin -flto=full 2> %t
// RUN: FileCheck -check-prefix=CHECK-LINK-FULL-ACTION < %t %s
//
// CHECK-LINK-FULL-ACTION: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"
// CHECK-LINK-FULL-ACTION-NOT: "-plugin-opt=thinlto"

// Check that subsequent -fno-lto takes precedence
// RUN: %clangxx -nocudainc -nocudalib \
// RUN:        -target x86_64-unknown-linux -### %s -flto=thin -fno-lto 2> %t
// RUN: FileCheck -check-prefix=CHECK-LINK-NOLTO-ACTION < %t %s
//
// CHECK-LINK-NOLTO-ACTION-NOT: "-plugin" "{{.*}}{{[/\\]}}LLVMgold.{{dll|dylib|so}}"
// CHECK-LINK-NOLTO-ACTION-NOT: "-plugin-opt=thinlto"
